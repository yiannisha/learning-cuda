#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define THREAD_COUNT 1024

// MxN * NxK matrix multiplication

void handle_err(hipError_t err) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
}

__global__
void matMulKernel(
    int* left,
    int* right,
    int* out,
    unsigned int M,
    unsigned int N,
    unsigned int K
)
{
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= M || col >= K) return;

    int outVal = 0;
    for (int i=0; i<N; i++) {
        outVal += left[row * N + i] * right[i * K + col];
    }
    out[row * K + col] = outVal;
}

int main()
{
    int length = 7;

    // MxN
    unsigned int length_A = 42;
    int A[length_A] = {
        69, 37, 17, 53, 91, 31, 74,
        40, 31, 75, 26, 30, 26, 29,
        47, 70, 1, 47, 12, 27, 21,
        17, 19, 56, 44, 53, 52, 40,
        13, 60, 49, 27, 30, 100, 81,
        98, 40, 51, 74, 94, 38, 38
    };
    unsigned int M = 6;

    // NxK
    unsigned int length_B = 21;
    int B[length_B] = {
        16,22,4,
        82,12,66,
        16,8,67,
        51,88,47,
        54,5,69,
        68,9,83,
        64,57,58
    };
    unsigned int K = 3;

    int *out_h, *out_d, *A_d, *B_d;
    unsigned int size = (M*K) * sizeof(int);
    out_h = (int *)malloc(size);

    hipError_t err;
    if ((err = hipMalloc((void**)&out_d, size)) != hipSuccess) handle_err(err);
    if ((err = hipMalloc((void**)&A_d, length_A*sizeof(int))) != hipSuccess) handle_err(err);
    if ((err = hipMalloc((void**)&B_d, length_B*sizeof(int))) != hipSuccess) handle_err(err);

    hipMemcpy(A_d, A, length_A*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, length_B*sizeof(int), hipMemcpyHostToDevice);

    // it is convenient to map 2D data to a 2D grid consisting of
    // 2D blocks
    dim3 dimGrid = dim3(ceil(K/(float)32), ceil(M/(float)32), 1);
    dim3 dimBlocks = dim3(32, 32, 1);

    matMulKernel<<<dimGrid, dimBlocks>>>(A_d, B_d, out_d, M, length, K);

    hipMemcpy(out_h, out_d, size, hipMemcpyDeviceToHost);

    for (int i=0; i<M; i++) {
        for (int j=0; j<K; j++) {
            printf("%d ", out_h[i*K + j]);
        }
        printf("\n");
    }

    hipFree(out_d);
    hipFree(A_d);
    hipFree(B_d);

    free(out_h);

    return EXIT_SUCCESS;
}